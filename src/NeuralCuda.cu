#include "hip/hip_runtime.h"
#include "NeuralCuda.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void FeedForwardKernel(float *inputs, float *weights, float *outputs, int inputSize, int outputSize, int weightsSize)
{
    int networkId = blockIdx.x;

    const int layerSizes[] = {32, 32, 32, 2};
    const int numLayers = sizeof(layerSizes) / sizeof(layerSizes[0]);

    int inputStartIndex = networkId * inputSize;
    int weightStartIndex = networkId * weightsSize;
    int outputStartIndex = networkId * outputSize;

    extern __shared__ float neuronValues[];

    for (int i = threadIdx.x; i < inputSize; i += blockDim.x)
    {
        neuronValues[i] = inputs[inputStartIndex + i];
    }
    __syncthreads();

    // Index to track the current position in the weights array
    int currentWeightIndex = weightStartIndex;

    for (int layer = 0; layer < numLayers; ++layer)
    {
        int numNeurons = layerSizes[layer];
        int prevLayerSize = (layer == 0) ? inputSize : layerSizes[layer - 1];

        if (threadIdx.x < numNeurons)
        {
            float sum = 0.0f;

            int weightIndex = currentWeightIndex + threadIdx.x * prevLayerSize;

            // Perform the weighted sum of inputs for this neuron
            for (int i = 0; i < prevLayerSize; ++i)
            {
                sum += weights[weightIndex + i] * neuronValues[i];
            }

            // Apply activation function (e.g., tanh)
            neuronValues[threadIdx.x] = tanh(sum);
        }

        // Update the currentWeightIndex for the next layer
        currentWeightIndex += numNeurons * prevLayerSize;

        __syncthreads();
    }

    if (threadIdx.x < outputSize)
    {
        outputs[outputStartIndex + threadIdx.x] = neuronValues[threadIdx.x];
    }
}

void CheckCudaError()
{
    hipError_t err;
    err = hipDeviceSynchronize();
    if (err != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", err);
        // Handle the error, e.g., free memory
    }
}

NeuralCuda::NeuralCuda(const std::vector<int> &layerSizes, int cars)
{
    this->amountOfCars = cars;
    this->layerSizes = layerSizes;
    this->numLayers = layerSizes.size();
    this->totalWeights = 0;
    for (size_t i = 0; i < layerSizes.size() - 1; i++)
    {
        this->totalWeights += layerSizes[i] * layerSizes[i + 1];
    }

    AllocateMemory();
}

NeuralCuda::~NeuralCuda()
{
    FreeMemory();
}

void NeuralCuda::AllocateMemory()
{
    // Allocate memory for inputs and outputs as before
    hipMalloc(&d_inputs, sizeof(float) * amountOfCars * layerSizes[0]);      // layerSizes[0] is the size of the input layer
    hipMalloc(&d_outputs, sizeof(float) * amountOfCars * layerSizes.back()); // layerSizes.back() is the size of the output layer

    // Calculate the total number of weights, excluding the input layer
    this->totalWeights = 0;
    for (int i = 0; i < numLayers - 1; i++)
    {
        this->totalWeights += layerSizes[i] * layerSizes[i + 1];
    }

    // Allocate memory for weights
    hipMalloc(&d_weights, sizeof(float) * this->totalWeights * amountOfCars);

    // Allocate and copy layerSizes
    hipMalloc(&d_layerSizes, sizeof(int) * numLayers);
    hipMemcpy(d_layerSizes, layerSizes.data(), sizeof(int) * numLayers, hipMemcpyHostToDevice);
}

void NeuralCuda::FreeMemory()
{
    hipFree(d_inputs);
    hipFree(d_weights);
    hipFree(d_outputs);
    hipFree(d_layerSizes);
}

void NeuralCuda::SetInputs(const std::vector<float> &inputs)
{
    hipMemcpy(d_inputs, inputs.data(), sizeof(float) * inputs.size(), hipMemcpyHostToDevice);
}

void NeuralCuda::SetWeights(const std::vector<float> &weights)
{
    hipMemcpy(d_weights, weights.data(), sizeof(float) * weights.size(), hipMemcpyHostToDevice);
    CheckCudaError();
}

void NeuralCuda::FeedForward(std::vector<float> &outputVec)
{
    // dim3 blockSize(1, 2); // 1 thread along x-axis (for 1 car), 2 threads along y-axis (for 2 outputs)
    // dim3 gridSize(1, 1);  // Only 1 block along both x and y axes for 1 car

    // Assuming the largest layer has 20 neurons
    int threadsPerBlock = 32;     // or a multiple of 32 for warp alignment, like 32
    int numBlocks = amountOfCars; // one block per network

    // Calculate shared memory size if used
    int sharedMemorySize = sizeof(float) * 32; // maxLayerSize is the size of the largest layer

    FeedForwardKernel<<<numBlocks, threadsPerBlock, sharedMemorySize>>>(d_inputs, d_weights, d_outputs, layerSizes[0], layerSizes.back(), this->totalWeights);

    // Assuming the output size is known and fixed (e.g., 300 * 2)
    std::vector<float> outputs(amountOfCars * 2);
    hipMemcpy(outputVec.data(), d_outputs, sizeof(float) * outputVec.size(), hipMemcpyDeviceToHost);

    CheckCudaError();
}

// Main function or other functions to interact with the NeuralNetwork class
