#include "hip/hip_runtime.h"
#include "CudaKernels.cuh"
#include <hiprand/hiprand_kernel.h>

__global__ void MatrixVectorMultiplyKernel(float *matrix, float *vector, float *output, int numRows, int numCols)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < numRows)
    {
        float sum = 0.0;
        for (int j = 0; j < numCols; ++j)
        {
            sum += matrix[row * numCols + j] * vector[j];
        }
        output[row] = sum;
    }
}

__global__ void ApplyTanhKernel(float *vector, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        vector[idx] = tanh(vector[idx]);
    }
}

__global__ void MatrixVectorMultiplyAndTanhKernel(float *matrix, float *vector, float *output, int numRows, int numCols)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < numRows)
    {
        float sum = 0.0;
        for (int j = 0; j < numCols; ++j)
        {
            sum += matrix[row * numCols + j] * vector[j];
        }
        output[row] = tanh(sum); // Apply tanh activation function here
    }
}

__global__ void MutateKernel(float *weights, int size, float mutationRate, float mutationScale, unsigned long long seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);

        if (hiprand_uniform(&state) < mutationRate)
        {
            weights[idx] += hiprand_normal(&state) * mutationScale;
        }
    }
}

__global__ void FullNetworkForwardPass(float *weights, float *neurons, int *layerSizes, int numLayers, float *output)
{
    int neuronId = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0;
    int weightIndex = 0;

    for (int layer = 1; layer < numLayers; ++layer)
    {
        if (neuronId < layerSizes[layer])
        {
            sum = 0.0;
            for (int j = 0; j < layerSizes[layer - 1]; ++j)
            {
                sum += weights[weightIndex + j] * neurons[j];
            }

            sum = tanh(sum); // Apply tanh activation function
            output[neuronId] = sum;

            weightIndex += layerSizes[layer - 1] * layerSizes[layer];
            neuronId += layerSizes[layer - 1];
        }
        __syncthreads(); // Synchronize threads at the end of each layer
    }
}