#include "NeuralNetworkCuda.h"
#include "CudaKernels.cuh"

NeuralNetworkCUDA::NeuralNetworkCUDA(const std::vector<int> &layers)
    : layers(layers), d_neurons(nullptr), d_weights(nullptr), fitness(0)
{
    AllocateDeviceMemory();
    InitNeurons();
    InitWeights();
}

NeuralNetworkCUDA::~NeuralNetworkCUDA()
{
    FreeDeviceMemory();
}

void NeuralNetworkCUDA::AllocateDeviceMemory()
{
    // Calculate total size for neurons and weights
    size_t totalNeuronsSize = 0;
    size_t totalWeightsSize = 0;
    for (size_t i = 0; i < layers.size(); ++i)
    {
        totalNeuronsSize += layers[i];
        if (i > 0)
        {
            totalWeightsSize += layers[i] * layers[i - 1];
        }
    }

    // Allocate memory on the device
    hipMalloc(&d_neurons, totalNeuronsSize * sizeof(float));
    hipMalloc(&d_weights, totalWeightsSize * sizeof(float));
}

void NeuralNetworkCUDA::FreeDeviceMemory()
{
    hipFree(d_neurons);
    hipFree(d_weights);
}

void NeuralNetworkCUDA::CopyToDevice()
{
    size_t offset = 0;
    for (size_t i = 0; i < layers.size(); ++i)
    {
        size_t layerSize = layers[i];

        // Copy neurons for each layer
        hipMemcpy(d_neurons + offset, host_neurons[i].data(), layerSize * sizeof(float), hipMemcpyHostToDevice);
        offset += layerSize;

        if (i > 0)
        {
            // Copy weights for each layer (excluding the input layer)
            size_t weightsSize = layers[i] * layers[i - 1];
            hipMemcpy(d_weights + offset, host_weights[i - 1].data(), weightsSize * sizeof(float), hipMemcpyHostToDevice);
            offset += weightsSize;
        }
    }
}

void NeuralNetworkCUDA::CopyToHost()
{
    // Example: Assuming you have a host-side representation of neurons
    size_t offset = 0;
    for (size_t i = 0; i < layers.size(); ++i)
    {
        size_t layerSize = layers[i];

        // Copy neurons for each layer
        hipMemcpy(host_neurons[i].data(), d_neurons + offset, layerSize * sizeof(float), hipMemcpyDeviceToHost);
        offset += layerSize;

        // If you need to copy weights back to host, do it similarly
        // ...
    }
}

void NeuralNetworkCUDA::InitNeurons()
{
    size_t totalNeuronsSize = 0;
    for (int layerSize : layers)
    {
        totalNeuronsSize += layerSize;
    }

    // Initialize neuron values to zero
    hipMemset(d_neurons, 0, totalNeuronsSize * sizeof(float));
}

void NeuralNetworkCUDA::InitWeights()
{
    size_t totalWeightsSize = 0;
    for (size_t i = 1; i < layers.size(); ++i)
    {
        totalWeightsSize += layers[i] * layers[i - 1];
    }

    // Host vector for weights
    std::vector<float> hostWeights(totalWeightsSize);

    // Initialize weights on the host
    for (float &weight : hostWeights)
    {
        weight = RandomWeight(); // Your method that returns a random float
    }

    // Copy initialized weights to the device
    hipMemcpy(d_weights, hostWeights.data(), totalWeightsSize * sizeof(float), hipMemcpyHostToDevice);
}

void NeuralNetworkCUDA::CopyWeights(const NeuralNetworkCUDA &copyFrom)
{
    // First, calculate the total size of all weights
    size_t totalWeightsSize = 0;
    for (size_t i = 1; i < layers.size(); ++i)
    {
        totalWeightsSize += layers[i] * layers[i - 1];
    }

    // Copy weights from one GPU memory space to another
    hipMemcpy(d_weights, copyFrom.d_weights, totalWeightsSize * sizeof(float), hipMemcpyDeviceToDevice);
}

std::vector<float> NeuralNetworkCUDA::FeedForward(const std::vector<float> &inputs)
{
    // Copy inputs to the device
    hipMemcpy(d_neurons, inputs.data(), inputs.size() * sizeof(float), hipMemcpyHostToDevice);
    float *d_input = d_neurons; // Initially, input is the first layer's neurons
    float *d_output;

    for (size_t i = 1; i < layers.size(); ++i)
    {
        int numInputNeurons = layers[i - 1];
        int numOutputNeurons = layers[i];

        // Allocate memory for the output of this layer
        hipMalloc(&d_output, numOutputNeurons * sizeof(float));

        // Set up grid and block dimensions for kernel
        int blockSize = 128; // A common choice, you might need to tune this
        int numBlocks = (numOutputNeurons + blockSize - 1) / blockSize;

        // Launch the kernel for this layer
        MatrixVectorMultiplyAndTanhKernel<<<numBlocks, blockSize>>>(d_weights, d_input, d_output, numOutputNeurons, numInputNeurons);

        // Wait for GPU to finish before continuing
        hipDeviceSynchronize();

        // Update d_input to point to the output of this layer
        // The output of this layer is the input to the next layer
        d_input = d_output;

        // Optionally, you can manage the memory more efficiently by reusing or freeing d_output
    }

    // Copy the final output back to host
    std::vector<float> output(layers.back());
    hipMemcpy(output.data(), d_output, output.size() * sizeof(float), hipMemcpyDeviceToHost);

    // Free the last d_output allocated inside the loop
    hipFree(d_output);

    return output;
}

void NeuralNetworkCUDA::Mutate(float mutationRate, float mutationScale)
{
    size_t totalWeightsSize = 0;
    for (size_t i = 1; i < layers.size(); ++i)
    {
        totalWeightsSize += layers[i] * layers[i - 1];
    }

    int blockSize = 256;
    int numBlocks = (totalWeightsSize + blockSize - 1) / blockSize;

    // Use a fixed seed for reproducibility, or vary it for different results
    unsigned long long seed = 1234;

    MutateKernel<<<numBlocks, blockSize>>>(d_weights, totalWeightsSize, mutationRate, mutationScale, seed);

    hipDeviceSynchronize();
}

float NeuralNetworkCUDA::GetFitness() const
{
    return fitness;
}

void NeuralNetworkCUDA::AddFitness(float fit)
{
    fitness += fit;
}

void NeuralNetworkCUDA::SetFitness(float fit)
{
    fitness = fit;
}

void NeuralNetworkCUDA::PrintWeights() const
{
    // Print weights (may require copying data back to host)
    // This is a placeholder for actual implementation
}

int NeuralNetworkCUDA::CompareTo(const NeuralNetworkCUDA &other) const
{
    return fitness > other.fitness ? 1 : (fitness < other.fitness ? -1 : 0);
}

float NeuralNetworkCUDA::RandomWeight()
{
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(-0.5f, 0.5f); // Example: normal distribution with mean 0 and stddev 1
    return dist(gen);
}
